#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
      
__global__ void matrixMultiply(float * A, float * B, float * C,
			       int numARows, int numAColumns,
			       int numBRows, int numBColumns,
			       int numCRows, int numCColumns) {
	
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    float temp = 0;

    if((Row < numCRows) && (Col < numCColumns)) { 

        for(int m = 0; m < numAColumns; ++m) {
            temp += A[Row * numAColumns + m] * B[m * numBColumns + Col];
        }
    
        C[Row * numCColumns + Col] = temp;
    }
}

int main(int argc, char ** argv) {
    float * hostA; // The A matrix
    float * hostB; // The B matrix
    float * hostC; // The output C matrix
    float * deviceA;
    float * deviceB;
    float * deviceC;
    int numARows = 4; // number of rows in the matrix A (Set this Value)
    int numAColumns = 3; // number of columns in the matrix A (Set this Value)
    int numBRows = 3; // number of rows in the matrix B (Set this Value)
    int numBColumns = 4; // number of columns in the matrix B (Set this Value)
    int numCRows; // number of rows in the matrix C
    int numCColumns; // number of columns in the matrix C


    // Set numCRows and numCColumns
    numCRows = numARows;
    numCColumns = numBColumns;
    // Allocate the hostC matrix
    int sizeA = numARows * numAColumns * sizeof(float);
    int sizeB = numBRows * numBColumns * sizeof(float);
    int sizeC = numCRows * numCColumns * sizeof(float);
    hostA = (float *) malloc(sizeA);
    hostB = (float *) malloc(sizeB);

    for(int i = 0; i < (numARows * numAColumns); i++) {
        hostA[i] = i;
    }

    for(int i = 0; i < (numBRows * numBColumns); i++) {
        hostB[i] = i;
    }

    hostC = (float *) malloc(sizeC);  
    // Allocate GPU memory here
    hipMalloc((void **) &deviceA, sizeA);
    hipMalloc((void **) &deviceB, sizeB);
    hipMalloc((void **) &deviceC, sizeC);

    // Copy memory to the GPU here

    hipMemcpy(deviceA, hostA, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(deviceB, hostB, sizeB, hipMemcpyHostToDevice);
    
    // Initialize the grid and block dimensions here
    
    dim3 dimGrid(ceil(((float)numCColumns)/16.0), ceil(((float) numCRows)/16.0), 1);
    dim3 dimBlock(16, 16, 1);
    matrixMultiply<<<dimGrid,dimBlock>>>(deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);

    hipDeviceSynchronize();
    
    // Copy the GPU memory back to the CPU here
    hipMemcpy(hostC, deviceC, sizeC, hipMemcpyDeviceToHost);

    for(int i = 0; i < (numCRows * numCColumns); i++) {
        printf("%f\n", hostC[i]);
    }
    // Free the GPU memory here
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);

    free(hostA);
    free(hostB);
    free(hostC);

    return 0;
}

